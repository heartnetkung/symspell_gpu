#include "test_util.cu"
#include "../src/cub.cu"

TEST(cal_combination_offset, {
	int inputLen = 4;
	char input[inputLen][7] = {"AC", "ACCC", "ACDEFG", "A"};
	Int3* input2;
	int* output;
	int distance = 2;
	int expected[] = {4, 15, 37, 39};

	hipMallocManaged(&input2, sizeof(Int3)*inputLen);
	hipMallocManaged(&output, sizeof(int)*inputLen);

	for (int i = 0; i < inputLen; i++)
		input2[i] = str_encode(input[i]);

	cal_combination_offset(input2, distance, output, inputLen);

	hipDeviceSynchronize();
	for (int i = 0; i < inputLen; i++)
		check(expected[i] == output[i]);

	hipFree(input2);
	hipFree(output);
})