#include "test_util.cu"
#include "../src/cub.cu"
#include "../src/kernel.cu"

TEST(cal_combination_offset, {
	int inputLen = 4;
	char input[inputLen][7] = {"AC", "ACCC", "ACDEFG", "A"};
	Int3* input2;
	int* input3;
	int* output;
	int distance = 2;
	int expected[] = {4, 15, 37, 39};

	hipMallocManaged(&input2, sizeof(Int3)*inputLen);
	hipMallocManaged(&input3, sizeof(int)*inputLen);
	hipMallocManaged(&output, sizeof(int)*inputLen);

	for (int i = 0; i < inputLen; i++)
		input2[i] = str_encode(input[i]);

	cal_combination_len <<< inputLen, 1>>>(input2, distance, input3, inputLen);
	inclusive_sum(input3, output, inputLen);

	hipDeviceSynchronize();
	for (int i = 0; i < inputLen; i++)
		check(expected[i] == output[i]);

	hipFree(input2);
	hipFree(input3);
	hipFree(output);
})

TEST(sort_pairs, {
	int inputLen = 4;
	char keys[inputLen][3] = {"AC", "AC", "A", "AC"};
	Int3* keys2;
	int* values;
	char expectedKeys[inputLen][3] = {"A", "AC", "AC", "AC"};
	int expectedValues[] = {5, 7, 6, 4};

	hipMallocManaged(&keys2, sizeof(int)*inputLen);
	hipMallocManaged(&values, sizeof(int)*inputLen);

	for (int i = 0; i < inputLen; i++) {
		keys2[i] = str_encode(keys[i]);
		values[i] = 7-i;
	}

	sort_pairs(keys2, values, inputLen);

	hipDeviceSynchronize();
	for (int i = 0; i < inputLen; i++) {
		checkstr(str_decode(keys2[i]), expectedKeys[i]);
		check(values[i] == expectedValues[i]);
	}

	hipFree(keys2);
	hipFree(values);
})