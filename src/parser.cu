#include "hip/hip_runtime.h"
#include "symspell.h"

// A CDEFGHI KLMN PQRST VW Y
const int A_char = (int)'A';
const int before_A_char = A_char - 1;
const int Y_char = (int) 'Y';

/**
 * encode character into 5 bit value (0-31).
 * -1 for non amino acid character
*/
int char_encode(char amino_acid) {
	if (amino_acid < A_char || amino_acid > Y_char)
		return -1;
	switch (amino_acid) {
	case 'B':
	case 'J':
	case 'O':
	case 'U':
	case 'X':
		return -1;
	default:
		return amino_acid - before_A_char;
	}
}

/**
 * encode peptide string into int3 struct with 6 characters encoded into an integer.
*/
Int3 str_encode(char *str) {
	Int3 ans;
	for (int i = 0; i < MAX_INPUT_LENGTH; i++) {
		char c = str[i];
		if (c == '\0')
			break; // end

		int value = char_encode(c);
		if (value == -1) {
			ans.entry[0] = 0;
			break; // invalid character
		}

		ans.entry[i / 6] |= value << (27 - 5 * (i % 6));
	}

	return ans;
}


/**
 * decode binary form into peptide string
*/
char* str_decode(Int3 binary) {
	char* ans = (char*) malloc((MAX_INPUT_LENGTH + 1) * sizeof(char));

	for (int i = 0; i < MAX_INPUT_LENGTH; i++) {
		char c = (binary.entry[i / 6] >> (27 - 5 * (i % 6))) & 0x1F;
		if (c == 0) {
			ans[i] = '\0';
			return ans;
		}

		ans[i] = before_A_char + c;
	}

	ans[MAX_INPUT_LENGTH] = '\0';
	return ans;
}