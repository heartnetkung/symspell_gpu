#include <stdio.h>
#include <stdlib.h>
#include "codec.cu"
#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"

const int NUM_THREADS = 256;

int gen_combinations(Int3* seq, int distance, Int3* &outputKeys, int* &outputValues, int n) {
	int *combinationOffsets;
	int seq1LenBlocks = divideCeil(n, NUM_THREADS);

	// cal combinationOffsets
	hipMalloc((void**)&combinationOffsets, sizeof(int)*n);
	gpuerr();
	cal_combination_len <<< seq1LenBlocks, NUM_THREADS >>>(
	    seq, distance, combinationOffsets, n);
	gpuerr();
	inclusive_sum(combinationOffsets, n);
	gpuerr();
	int outputLen = transfer_last_element(combinationOffsets, n);
	gpuerr();

	// generate combinations
	hipMalloc(&outputKeys, sizeof(Int3)*outputLen);
	gpuerr();
	hipMalloc(&outputValues, sizeof(int)*outputLen);
	gpuerr();
	gen_combination <<< seq1LenBlocks, NUM_THREADS >>> (
	    seq, combinationOffsets, distance, outputKeys, outputValues, n);
	gpuerr();

	hipFree(combinationOffsets);
	gpuerr();
	return outputLen;
}

int cal_offsets(Int3* inputKeys, int* inputValues, int* &inputOffsets, int* &outputLengths, int n, int* buffer) {
	// cal valueOffsets
	hipMalloc(&inputOffsets, sizeof(int)*n);
	gpuerr();
	sort_key_values(inputKeys, inputValues, n);
	gpuerr();
	unique_counts(inputKeys, inputOffsets, buffer, n);
	gpuerr();

	// cal pairOffsets
	int nUnique = transfer_last_element(buffer, 1);
	gpuerr();
	int nUniqueBlock = divideCeil(nUnique, NUM_THREADS);
	hipMalloc(&outputLengths, sizeof(int)*nUnique);
	gpuerr();
	cal_pair_len <<< nUniqueBlock, NUM_THREADS>>>(inputOffsets, outputLengths, nUnique);
	gpuerr();
	inclusive_sum(inputOffsets, nUnique);
	gpuerr();
	return nUnique;
}

//inputOffsets, outputLengths, n moved as per loop
int gen_pairs(int* input, int* inputOffsets, int &carry, int* outputLengths, Int2* &output, int n, int* buffer) {
	// generate output offsets
	int* outputOffsets;
	hipMalloc(&outputOffsets, sizeof(int)*n);
	gpuerr();
	inclusive_sum(outputLengths, outputOffsets, n);
	gpuerr();

	// generate pairs
	int outputLen = transfer_last_element(outputOffsets, n);
	gpuerr();
	int nBlock = divideCeil(n, NUM_THREADS);
	hipMalloc(&output, sizeof(Int2)*outputLen);
	gpuerr();
	generate_pairs <<< nBlock, NUM_THREADS>>>(input, carry, output, inputOffsets, outputOffsets, n);
	gpuerr();

	carry += transfer_last_element(inputOffsets, n);
	gpuerr();
	hipFree(outputOffsets);
	gpuerr();
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance,
                   Int2* &pairOutput, char* &distanceOutput,
                   int n, int* buffer, int seqLen) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n);
	gpuerr();
	sort_int2(input, n);
	gpuerr();
	unique(input, uniquePairs, buffer, n);
	gpuerr();

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 1);
	int byteRequirement = sizeof(char) * uniqueLen;
	int uniqueLenBlock = divideCeil(uniqueLen, NUM_THREADS);
	hipMalloc(&flags, byteRequirement);
	gpuerr();
	hipMalloc(&uniqueDistances, byteRequirement);
	gpuerr();
	hipMalloc(&distanceOutput, byteRequirement);
	gpuerr();
	hipMalloc(&pairOutput, sizeof(Int2)*uniqueLen);
	gpuerr();
	cal_levenshtein <<< uniqueLenBlock, NUM_THREADS>>>(
	    seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen, seqLen);
	gpuerr();

	// filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput,
	            distanceOutput, buffer, uniqueLen);
	gpuerr();

	_cudaFree(uniquePairs, uniqueDistances, flags);
	gpuerr();
	return transfer_last_element(buffer, 1);
}

int concat_buffers(Int2** keyBuffer, char** valueBuffer, int* bufferLengths,
                   Int2* &keyOutput, char* &valueOutput, int n) {
	int totalBufferLength = 0;
	for (int i = 0; i < n; i++)
		totalBufferLength += bufferLengths[i];

	hipMalloc(&keyOutput, sizeof(Int2)*totalBufferLength);
	gpuerr();
	hipMalloc(&valueOutput, sizeof(char)*totalBufferLength);
	gpuerr();

	Int2* keyOutputP = keyOutput;
	char* valueOutputP = valueOutput;
	int bufferLength;
	for (int i = 0; i < n; i++) {
		bufferLength = bufferLengths[i];
		hipMemcpy(keyOutputP, keyBuffer[i], sizeof(Int2)*bufferLength, hipMemcpyDeviceToDevice);
		gpuerr();
		hipMemcpy(valueOutputP, valueBuffer[i], sizeof(char)*bufferLength, hipMemcpyDeviceToDevice);
		gpuerr();
		keyOutputP += bufferLength;
		valueOutputP += bufferLength; // divided by 4?
	}

	return totalBufferLength;
}

int remove_duplicate(Int2* keyInput, char* valueInput, Int2* &keyOutput, char* &valueOutput, int n, int* buffer) {
	char* flags;
	int* runOffsets, *runLengths;

	hipMalloc(&keyOutput, sizeof(Int2)*n);
	gpuerr();
	hipMalloc(&valueOutput, sizeof(char)*n);
	gpuerr();
	hipMalloc(&flags, sizeof(char)*n);
	gpuerr();
	hipMalloc(&runOffsets, sizeof(int)*n);
	gpuerr();
	hipMalloc(&runLengths, sizeof(int)*n);
	gpuerr();

	// sort
	sort_key_values2(keyInput, valueInput, n);
	gpuerr();

	// make flag
	non_trivial_runs(keyInput, runOffsets, runLengths, buffer, n);
	gpuerr();
	int runLength = transfer_last_element(buffer, 1);
	gpuerr();
	hipMemset(flags, 1, sizeof(char)*n);
	gpuerr();
	int runBlock = divideCeil(runLength, NUM_THREADS);
	non_trivial_runs_flag <<< runBlock, NUM_THREADS>>>(runOffsets, runLengths, flags, runLength);
	gpuerr();

	//filter
	double_flag(keyInput, valueInput, flags, keyOutput, valueOutput, buffer, n);
	gpuerr();

	_cudaFree(flags, runOffsets, runLengths);
	gpuerr();
	return transfer_last_element(buffer, 1);
}

int symspell_perform(SymspellArgs args, Int3* seq1, SymspellOutput* output) {
	int distance = args.distance, verbose = args.verbose, seq1Len = args.seq1Len, nSegment = args.nSegment;
	int* deviceInt;
	hipMalloc((void**)&deviceInt, sizeof(int));

	//=====================================
	// step 1: transfer input to GPU
	//=====================================
	Int3* seq1Device = host_to_device(seq1, seq1Len);
	print_tp(verbose, "1", seq1Len);

	//=====================================
	// step 2: generate deletion combinations
	//=====================================
	Int3* combinationKeys;
	int* combinationValues;
	int combinationLen =
	    gen_combinations(seq1Device, distance, combinationKeys, combinationValues, seq1Len);

	print_tp(verbose, "2", combinationLen);

	//=====================================
	// step 3: calculate pair offsets from combination values
	//=====================================
	int* combinationValueOffsets, *pairLengths;
	int offsetLen =
	    cal_offsets(combinationKeys, combinationValues, combinationValueOffsets,
	                pairLengths, combinationLen, deviceInt);

	print_tp(verbose, "3", offsetLen);
	hipFree(combinationKeys);

	//=====================================
	// step 4: generate output buffers segment by segment
	//=====================================
	Int2** pairBuffer = (Int2**)calloc(nSegment,sizeof(Int2*));
	char** distanceBuffer = (char**)calloc(nSegment,sizeof(char*));
	int* bufferLengths = (int*)calloc(nSegment,sizeof(int));

	int chunkPerSegment = divideCeil(offsetLen, nSegment);
	Int2* tempPairs;
	int tempPairLength;
	int carry = 0;
	int *pairLengthsP = pairLengths, *combinationValueOffsetsP = combinationValueOffsets;

	for (int i = 0; i < nSegment; i++) {
		// the last segment can be smaller than others
		if ((i == nSegment - 1) && (nSegment != 1)) {
			if (offsetLen % chunkPerSegment != 0)
				chunkPerSegment = offsetLen % chunkPerSegment;
		}

		tempPairLength =
		    gen_pairs(combinationValues, combinationValueOffsetsP, carry,
		              pairLengthsP, tempPairs, chunkPerSegment, deviceInt);
		bufferLengths[i] =
		    postprocessing(seq1Device, tempPairs, distance, pairBuffer[i],
		                   distanceBuffer[i], tempPairLength, deviceInt, seq1Len);
		print_tp(verbose, "4.1", tempPairLength);
		print_tp(verbose, "4.2", bufferLengths[i]);

		combinationValueOffsetsP += chunkPerSegment;
		pairLengthsP += chunkPerSegment;

		hipFree(tempPairs);
	}

	_cudaFree(seq1Device, combinationValues, combinationValueOffsets, pairLengths);

	//=====================================
	// step 5: merge buffers
	//=====================================
	Int2* outputPairs;
	char* outputDistances;
	int outputLen;

	if (nSegment == 1) {
		outputPairs = pairBuffer[0];
		outputDistances = distanceBuffer[0];
		outputLen = bufferLengths[0];
	} else {
		Int2* pairAllBuffer;
		char* distanceAllBuffer;
		int allBufferLen = concat_buffers(pairBuffer, distanceBuffer, bufferLengths,
		                                  pairAllBuffer, distanceAllBuffer, nSegment);
		print_tp(verbose, "5.1", allBufferLen);

		outputLen = remove_duplicate(
		                pairAllBuffer, distanceAllBuffer, outputPairs,
		                outputDistances, allBufferLen, deviceInt);
		_cudaFree(pairAllBuffer, distanceAllBuffer);
	}

	print_tp(verbose, "5", outputLen);

	//=====================================
	// step 6: transfer output to CPU
	//=====================================
	output->indexPairs = device_to_host(outputPairs, outputLen);
	output->pairwiseDistances = device_to_host(outputDistances, outputLen);
	output->len = outputLen;

	print_tp(verbose, "6", outputLen);
	_cudaFree(deviceInt, outputPairs, outputDistances);
	for (int i = 0; i < nSegment; i++)
		_cudaFree(pairBuffer[i], distanceBuffer[i]);
	_free(pairBuffer, distanceBuffer, bufferLengths);
	return 0;
}

void symspell_free(SymspellOutput *output) {
	if (output->indexPairs) {
		hipHostFree(output->indexPairs);
		output->indexPairs = NULL;
	}
	if (output->pairwiseDistances) {
		hipHostFree(output->pairwiseDistances);
		output->pairwiseDistances = NULL;
	}
}