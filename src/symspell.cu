#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "codec.cu"
#include "generate_combination.cu"
#include "cub.cu"
#include "kernel.cu"

const int NUM_THREADS = 256;

int gen_combinations(Int3* seq, int distance, Int3* outputKeys, int* outputValues, int n) {
	int *combinationOffsets;
	int seq1LenBlocks = (int)ceil(n / NUM_THREADS);

	// cal combinationOffsets
	hipMalloc((void**)&combinationOffsets, sizeof(int)*n);
	cal_combination_len <<< seq1LenBlocks, NUM_THREADS >>>(
	    seq, distance, combinationOffsets, n);
	inclusive_sum(combinationOffsets, n);
	int outputLen = transfer_last_element(combinationOffsets, n);

	// generate combinations
	hipMalloc((void**)&outputKeys, sizeof(Int3)*outputLen);
	hipMalloc((void**)&outputValues, sizeof(int)*outputLen);
	gen_combination <<< seq1LenBlocks, NUM_THREADS >>> (
	    seq, combinationOffsets, distance, outputKeys, outputValues, n);

	hipFree(combinationOffsets);
	return outputLen;
}

int gen_pairs(Int3* inputKeys, int* inputValues, Int2* output, int n, int* buffer) {
	int* valueOffsets, *pairOffsets;

	// cal valueOffsets
	hipMalloc(&valueOffsets, sizeof(int)*n);
	sort_key_values(inputKeys, inputValues, n);
	unique_counts(inputKeys, valueOffsets, buffer, n);

	// cal pairOffsets
	int nUnique = transfer_last_element(buffer, 0);
	int nUniqueBlock = (int)ceil(n / NUM_THREADS);
	hipMalloc(&pairOffsets, sizeof(int)*nUnique);
	cal_pair_len <<< nUniqueBlock, NUM_THREADS>>>(valueOffsets, pairOffsets, nUnique);
	inclusive_sum(valueOffsets, nUnique);
	inclusive_sum(pairOffsets, nUnique);

	// generate pairs
	int outputLen = transfer_last_element(pairOffsets, nUnique);
	hipMallocManaged(&output, sizeof(Int2)*outputLen);
	generate_pairs <<< nUniqueBlock, NUM_THREADS>>>(values, output, valueOffsets, pairOffsets, nUnique);

	_cudaFree(valueOffsets, pairOffsets);
	return outputLen;
}

int postprocessing(Int3* seq, Int2* input, int distance, Int2* pairOutput, char* distanceOutput, int n, int* buffer) {
	Int2* uniquePairs;
	char* uniqueDistances, *flags;

	// filter duplicate
	hipMalloc(&uniquePairs, sizeof(Int2)*n);
	sort_int2(input, n);
	unique(input, uniquePairs, buffer, n);

	// cal levenshtein
	int uniqueLen = transfer_last_element(buffer, 0);
	int byteRequirement = sizeof(char) * uniqueLen;
	hipMalloc(&flags, byteRequirement);
	hipMalloc(&uniqueDistances, byteRequirement);
	hipMalloc(&distanceOutput, byteRequirement);
	cal_levenshtein(seq, uniquePairs, distance, uniqueDistances, flags, uniqueLen);

	//filter levenshtein
	double_flag(uniquePairs, uniqueDistances, flags, pairOutput, distanceOutput, buffer, uniqueLen);

	_cudaFree(uniquePairs, uniqueDistances, flags);
	return transfer_last_element(buffer, 0);
}

int symspell_perform(SymspellArgs args, Int3* seq1, SymspellOutput* output) {
	int distance = args.distance, verbose = args.verbose, seq1Len = args.seq1Len;
	int* deviceInt;
	hipMalloc((void**)&deviceInt, sizeof(int));

	//=====================================
	// step 1: transfer input to GPU
	//=====================================
	Int3* seq1Device;
	int seq1Bytes = sizeof(Int3) * seq1Len;

	hipMalloc((void**)&seq1Device, seq1Bytes);
	hipMemcpy(seq1Device, seq1, seq1Bytes, hipMemcpyHostToDevice);

	if (verbose)
		printf("step 1 completed\n");

	//=====================================
	// step 2: generate deletion combinations
	//=====================================
	Int3* combinationKeys;
	int* combinationValues;
	int combinationLen =
	    gen_combinations(seq1Device, distance, combinationKeys, combinationValues, seq1Len);

	if (verbose)
		printf("step 2 completed\n");

	//=====================================
	// step 3: turn combinations into pairs
	//=====================================
	Int2* pairs;
	int pairLength =
	    gen_pairs(combinationKeys, combinationValues, pairs, combinationLen, deviceInt);

	if (verbose)
		printf("step 3 completed\n");

	//=====================================
	// step 4: Levenshtein/duplicate postprocessing
	//=====================================
	Int2* outputPairs;
	char* outputDistances;
	int outputLen =
	    postprocessing(seq1Device, pairs, distance, outputPairs, outputDistances, pairLength, deviceInt);

	if (verbose)
		printf("step 4 completed\n");

	//=====================================
	// step 5: transfer output to CPU
	//=====================================
	int pairBytes = sizeof(Int2) * outputLen;
	hipHostMalloc((void**)&output->indexPairs, pairBytes);
	hipMemcpy(outputPairs, output->indexPairs, pairBytes, hipMemcpyDeviceToHost);

	int distanceBytes = sizeof(char) * outputLen;
	hipHostMalloc((void**)&output->pairwiseDistances, distanceBytes);
	hipMemcpy(outputDistances, output->pairwiseDistances, distanceBytes, hipMemcpyDeviceToHost);

	output->len = outputLen;

	if (verbose)
		printf("step 5 completed\n");

	hipDeviceSynchronize();
	_cudaFree(deviceInt, seq1Device, combinationKeys, combinationValues, pairs, outputPairs, outputDistances);
	return 0;
}

void symspell_free(SymspellOutput* output) {
	if (output->indexPairs) {
		hipHostFree(output->indexPairs);
		output->indexPairs = NULL;
	}
	if (output->pairwiseDistances) {
		hipHostFree(output->pairwiseDistances);
		output->pairwiseDistances = NULL;
	}
}

