#include <cub/device/device_scan.cuh>
#include <cub/device/device_merge_sort.cuh>
#include "codec.cu"

struct Int3Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int3 &lhs, const Int3 &rhs) {
		if (lhs.entry[0] != rhs.entry[0])
			return lhs.entry[0] < rhs.entry[0];
		if (lhs.entry[1] != rhs.entry[1])
			return lhs.entry[0] < rhs.entry[0];
		return lhs.entry[2] < rhs.entry[2];
	}
};

struct Int2Comparator {
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	bool operator()(const Int2 &lhs, const Int2 &rhs) {
		if (lhs.x != rhs.x)
			return lhs.x < rhs.x;
		return lhs.y < rhs.y;
	}
};

void inclusive_sum(int* input, int* output, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceScan::InclusiveSum(buffer, bufferSize, input, output, n);
	hipFree(buffer);
}

void sort_key_values(Int3* keys, int* values, int n) {
	void *buffer = NULL;
	size_t bufferSize = 0;
	Int3Comparator op;
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);
	hipMalloc(&buffer, bufferSize);
	hipcub::DeviceMergeSort::SortPairs(buffer, bufferSize, keys, values, n, op);
	hipFree(buffer);
}
