#include "symspell.h"

int print_err(const char* str) {
#ifndef TEST_ENV
	fprintf(stderr, "Error: %s\n", str);
#endif
	return ERROR;
}

void print_int3(Int3* seqs, int len, char prefix) {
	int n_elements = len < 5 ? len : 5;
	for (int i = 0; i < n_elements; i++) {
		unsigned int* entry = seqs[i].entry;
		printf("%c %08X %08X %08X \n", prefix, entry[0], entry[1], entry[2]);
	}
}

void print_args(SymspellArgs args) {
	printf("SymspellArgs{\n");
	printf("\tdistance: %d\n", args.distance);
	printf("\tverbose: %d\n", args.verbose);
	printf("\tseq1Len: %d\n", args.seq1Len);
	printf("\tseq1Path: \"%s\"\n", args.seq1Path);
	printf("}\n");
}

void print_int_arr(int* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
}

void print_char_arr(char* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
}

void print_int2_arr(Int2* arr, int n) {
	for (int i = 0; i < n; i++)
		printf("(%d,%d) ", arr[i].x, arr[i].y);
	printf("\n");
}

void _cudaFree(void* a) {
	hipFree(a);
}
void _cudaFree(void* a, void* b) {
	hipFree(a);
	hipFree(b);
}
void _cudaFree(void* a, void* b, void* c) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
}
void _cudaFree(void* a, void* b, void* c, void* d) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(f);
}
void _cudaFree(void* a, void* b, void* c, void* d, void* e, void* f, void* g) {
	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
	hipFree(e);
	hipFree(g);
}